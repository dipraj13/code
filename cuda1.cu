#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int* b, int* c,int n){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < n){
		c[id] = a[id] + b[id];
	}
}

int main(){
	int n = 1 << 20;
	int size = n * sizeof(int);
	
	int *h1 = (int*) malloc(size);
	int *h2 = (int*) malloc(size);
	int *h3 = (int*) malloc(size);
	
	for(int i =0;i<n;i++){
		h1[i] = i;
		h2[i] = 2* i;
	}
	int *d1,*d2,*d3;
	
	hipMalloc((void**)&d1,size);
	hipMalloc((void**)&d2,size);
	hipMalloc((void**)&d3,size);
	
	hipMemcpy(d1,h1,size,hipMemcpyHostToDevice);
	hipMemcpy(d2,h2,size,hipMemcpyHostToDevice);
	
	add<<<(n+255)/256,256>>>(d1,d2,d3,n);
	
	hipMemcpy(h3,d3,size,hipMemcpyDeviceToHost);
	
	printf("Samp Out : %d + %d = %d \n",h1[12],h2[12],h3[12]);
	
	hipFree(d1);
	hipFree(d2);
	hipFree(d3);
	
	free(h1);
	free(h2);
	free(h3);
	
	return 0;
		
}
