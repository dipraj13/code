#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 16

__global__ void mul(int *a,int *b,int *c,int width){
	int rows = threadIdx.y + blockIdx.y * blockDim.y;
	int cols = threadIdx.x + blockIdx.x * blockDim.x;
	
	int sum = 0;

	if(rows < width && cols < width){
		for(int k = 0;k < width;k++)
			sum += a[rows * width + k] * b[k * width + cols];
		c[rows * width + cols] = sum;
	}	
}

int main(){
	int size = N * N *sizeof(int);
	int h1[N*N], h2[N*N], h3[N*N];
	
	for(int i = 0;i < N*N;i++){
		h1[i] = 1;
		h2[i] = 2;
	}
	
	int *d1,*d2,*d3;
	
	hipMalloc((void**)&d1,size);
	hipMalloc((void**)&d2,size);
	hipMalloc((void**)&d3,size);
	
	hipMemcpy(d1,h1,size,hipMemcpyHostToDevice);
	hipMemcpy(d2,h2,size,hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(16,16);
	dim3 blocksPerGrid((N+15)/16,(N+15)/16);
	
	mul<<<blocksPerGrid, threadsPerBlock>>>(d1,d2,d3,N);
	
	hipMemcpy(h3,d3,size,hipMemcpyDeviceToHost);
	
	printf("Output : %d",h2[12]);
	
	hipFree(d1);
	hipFree(d2);
	hipFree(d3);
	
	return 0;
}
